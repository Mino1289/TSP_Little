/**
* Projec : gtsp (voyageur de commerce)
*
* Date   : 07/04/2014
* Author : Olivier Grunder
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <stdbool.h>

// #define LITTLEPLUS
// #ifdef LITTLEPLUS
// #warning "LITTLEPLUS is defined"
// #else
// #warning "LITTLEPLUS is not defined"
// #endif

#define BIG_VALUE 1e9
#define NBR_TOWNS 10

/* Distance matrix */
float dist[NBR_TOWNS][NBR_TOWNS];

/* next_town[i] = next town after town i */
int next_town[NBR_TOWNS];

/* no comment */
int best_solution[NBR_TOWNS];
float best_eval = -1.0;


/**
* Berlin52 :
*  6 towns : (2608.25) -> Best solution (2315.15): 0 1 2 3 5 4
* 10 towns : (3278.83) -> Best solution (2826.50): 0 1 6 2 7 8 9 3 5 4
* 15 towns : (5075.43) -> Best solution (4967.300): 0 13 12 10 11 3 5 4 14 9 8 7 2 6 1
* 20 towns : (5637.21) -> Best solution (5270.860): 0 17 2 16 6 1 19 15 13 12 10 11 3 5 4 14 9 8 7 18
* 25 towns : (5773.28) -> Best solution (5460.939): 0 21 17 20 1 6 16 2 18 7 8 9 14 4 23 5 3 24 11 10 12 13 15 19 22
* 30 towns : (6875.88) -> Best solution ? 
* 35 towns : (6888.63) -> Best solution ?
* 40 towns : (7000.24) -> Best solution ?
* 45 towns : (7549.86) -> Best solution ?
* 50 towns : (8804.23) -> Best solution ?
* 52 towns : (8980.91) -> Best solution ?
* A280 :
* 280 towns : (4148.110) -> Best solution ?
* eil76:
* 76 towns : (711.993) -> Best solution ?
* KroA100 :
* 100 towns : (26856.389) -> Best solution ?
*/

// Berlin52
float coord[NBR_TOWNS][2] =
{
    {565.0, 575.0},
    {25.0, 185.0},
    {345.0, 750.0},
    {945.0, 685.0},
    {845.0, 655.0},
    {880.0, 660.0},
    {25.0, 230.0},
    {525.0, 1000.0},
    {580.0, 1175.0},
    {650.0, 1130.0},
    // {1605.0, 620.0}, 
    // {1220.0, 580.0},
    // {1465.0, 200.0},
    // {1530.0, 5.0},
    // {845.0, 680.0},
    // {725.0, 370.0},
    // {145.0, 665.0},
    // {415.0, 635.0},
    // {510.0, 875.0},  
    // {560.0, 365.0},
    // {300.0, 465.0},
    // {520.0, 585.0},
    // {480.0, 415.0},
    // {835.0, 625.0},
    // {975.0, 580.0},
    // {1215.0, 245.0},
    // {1320.0, 315.0},
    // {1250.0, 400.0},
    // {660.0, 180.0},
    // {410.0, 250.0},
    // {420.0, 555.0},
    // {575.0, 665.0},
    // {1150.0, 1160.0},
    // {700.0, 580.0},
    // {685.0, 595.0},
    // {685.0, 610.0},
    // {770.0, 610.0},
    // {795.0, 645.0},
    // {720.0, 635.0},
    // {760.0, 650.0},
    // {475.0, 960.0},
    // {95.0, 260.0},
    // {875.0, 920.0},
    // {700.0, 500.0},
    // {555.0, 815.0},
    // {830.0, 485.0},
    // {1170.0, 65.0},
    // {830.0, 610.0},
    // {605.0, 625.0},
    // {595.0, 360.0},
    // {1340.0, 725.0},
    // {1740.0, 245.0},
};



/**
* print a matrix
*/
void print_matrix(float d[NBR_TOWNS][NBR_TOWNS]) {
    int i, j;
    for (i = 0; i < NBR_TOWNS; i++) {
        printf("%d\t", i + 1);
        for (j = 0; j < NBR_TOWNS; j++) {
            printf("%.3f ", d[i][j]);
        }
        printf("\n");
    }
}



/**
* compute distance matrix
*/
void compute_matrix(float d[NBR_TOWNS][NBR_TOWNS]) {
    int i, j;
    for (i = 0; i < NBR_TOWNS; i++) {
        float xi = coord[i][0];
        float yi = coord[i][1];

        for (j = 0; j < NBR_TOWNS; j++) {
            if (i == j) {
                d[i][j] = -1;
                continue;
            }
            float xj = coord[j][0];
            float yj = coord[j][1];
            d[i][j] = sqrt((xi - xj) * (xi - xj) + (yi - yj) * (yi - yj));
        }
    }
}

/**
* print a solution
*/
void print_solution(int* sol, float eval) {
    int i;
    printf("(%.3f): ", eval);
    for (i = 0; i < NBR_TOWNS; i++)
        printf("%d ", sol[i]);
    printf("\n");
}


/**
* evaluation of a solution
*/
float evaluation_solution(int* sol) {
    float eval = 0;
    int i;
    int start, end;
    for (i = 0; i < NBR_TOWNS; i++) {
        start = sol[i];
        end = sol[(i + 1) % NBR_TOWNS];
        eval += dist[start][end];
    }

    return eval;
}

int find_nearest_unvisited(int current_town, bool visited[NBR_TOWNS]) {
    int nearest_town = -1;
    float min_distance = -1;

    for (int j = 0; j < NBR_TOWNS; j++) {
        if (!visited[j]) {
            float distance = dist[current_town][j];
            if (min_distance < 0 || distance < min_distance) {
                min_distance = distance;
                nearest_town = j;
            }
        }
    }
    return nearest_town;
}


/**
* initial solution
*/
float initial_solution() {
    /* solution of the nearest neighbour */
    int sol[NBR_TOWNS];
    bool visited[NBR_TOWNS] = { false }; // Initialize all towns as unvisited
    float eval = 0.0;

    sol[0] = 0;
    visited[0] = true;

    for (int i = 1; i < NBR_TOWNS; i++) {
        int current_town = sol[i - 1];
        int next_town = find_nearest_unvisited(current_town, visited);

        sol[i] = next_town;
        visited[next_town] = true;
    }

    eval = evaluation_solution(sol);
    printf("Initial solution ");
    print_solution(sol, eval);

    /* initialize best solution with initial solution */
    for (int i = 0; i < NBR_TOWNS; i++)
        best_solution[i] = sol[i];
    best_eval = eval;

    return eval;
}




/**
*  Build final solution
*/
void build_solution() {
    int i, solution[NBR_TOWNS];

    int indiceCour = 0;
    int villeCour = 0;

    for (indiceCour = 0; indiceCour < NBR_TOWNS; indiceCour++) {

        solution[indiceCour] = villeCour;

        /* Test si le cycle est hamiltonien */
        for (i = 0; i < indiceCour; i++) {
            if (solution[i] == villeCour) {
                // printf ("cycle non hamiltonien\n");
                return;
            }
        }
        /* Recherche de la ville suivante */
        villeCour = next_town[villeCour];
    }

    float eval = evaluation_solution(solution);

    if (best_eval < 0 || eval < best_eval) {
        best_eval = eval;
        for (i = 0; i < NBR_TOWNS; i++)
            best_solution[i] = solution[i];

        printf("New best solution: ");
        print_solution(solution, best_eval);
    }
    return;
}



/* Check if adding an edge creates a sub-tour */
bool creates_subtour(int start, int end) {
    int tmp = next_town[start];
    int i = 0;

    while (tmp != end && tmp != -1 && i < NBR_TOWNS) {
        i++;
        tmp = next_town[tmp];
        // if (tmp == end)
        //     return true;
    }

    if (tmp == start && i < NBR_TOWNS) {
        return true;
    }
    return false;
}

__global__ void min_rows(float d[NBR_TOWNS*NBR_TOWNS], float eval_node) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j;

    float minl = -1;
    for (j = 0; j < NBR_TOWNS; j++) {
        float val = d[i * NBR_TOWNS + j];
        if (minl < 0 || (val >= 0 && val < minl)) {
            minl = val;
        }
    }
    if (minl >= 0) {
        eval_node += minl;
        for (j = 0; j < NBR_TOWNS; j++) {
            if (d[i * NBR_TOWNS + j] >= 0) {
                d[i * NBR_TOWNS + j] -= minl;
            }
        }
    }
}

__global__ void min_col(float d[NBR_TOWNS * NBR_TOWNS], float eval_node) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i;

    float minc = -1;
    for (i = 0; i < NBR_TOWNS; i++) {
        float val = d[i * NBR_TOWNS + j];
        if (minc < 0 || (val >= 0 && val < minc)) {
            minc = val;
        }
    }
    if (minc >= 0) {
        eval_node += minc;
        for (i = 0; i < NBR_TOWNS; i++) {
            if (d[i * NBR_TOWNS + j] >= 0) {
                d[i * NBR_TOWNS + j] -= minc;
            }
        }
    }
    
}

__global__ void find_zeros(float d[NBR_TOWNS * NBR_TOWNS], float* penaltys) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int k;

    if (d[i * NBR_TOWNS + j] == 0) {
        float min_row = -1;
        float min_col = -1;

        for (k = 0; k < NBR_TOWNS; k++) {
            float valik = d[i * NBR_TOWNS + k];
            float valkj = d[k * NBR_TOWNS + j];

            if (k != j && valik >= 0 && (min_row < 0 || valik < min_row)) {
                min_row = valik;
            }
            if (k != i && valkj >= 0 && (min_col < 0 || valkj < min_col)) {
                min_col = valkj;
            }
        }
        if (min_row < 0 || min_col < 0) {
            min_row = BIG_VALUE;
            min_col = BIG_VALUE;
        }
        penaltys[i * NBR_TOWNS + j] = min_row + min_col;
    }
}


/**
*  Little Algorithm
*/
void little_algorithm(float d0[NBR_TOWNS][NBR_TOWNS], int iteration, float eval_node_parent) {
    if (iteration == NBR_TOWNS) {
        build_solution();
        return;
    }

    /* Do the modification on a copy of the distance matrix */
    // float d[NBR_TOWNS][NBR_TOWNS];
    float *d;
    hipMallocManaged(&d, NBR_TOWNS * NBR_TOWNS * sizeof(float));
    memcpy(d, d0, NBR_TOWNS * NBR_TOWNS * sizeof(float));

    float eval_node_child = eval_node_parent;

    dim3 threads_per_block (4, 4, 1); // A 16 x 1 block threads
    dim3 number_of_blocks ((NBR_TOWNS / threads_per_block.x) + 1, 1, 1);


    min_rows<<< number_of_blocks, threads_per_block >>>(d, eval_node_child);
    hipDeviceSynchronize(); // Wait for the GPU to finish before proceeding
    min_col<<< number_of_blocks, threads_per_block >>>(d, eval_node_child);
    hipDeviceSynchronize(); // Wait for the GPU to finish before proceeding


    /* Cut : stop the exploration of this node */
    if (best_eval >= 0 && eval_node_child >= best_eval) {
        return;
    }

    /* row and column of the zero with the max penalty */
    int izero = -1, jzero = -1;
    float max_penalty = -1;
    int i,j,k;


    float *penaltys;
    hipMallocManaged(&penaltys, NBR_TOWNS * NBR_TOWNS * sizeof(float));

    dim3 threads_per_block2 (4, 4, 1); // A 16 x 16 block threads
    dim3 number_of_blocks2 ((NBR_TOWNS / threads_per_block2.x) + 1, (NBR_TOWNS / threads_per_block2.y) + 1, 1);

    find_zeros<<< number_of_blocks2, threads_per_block2 >>>(d, penaltys);
    hipDeviceSynchronize(); // Wait for the GPU to finish before proceeding

    max_penalty = -1;
    for (i = 0; i < NBR_TOWNS; i++) {
        for (j = 0; j < NBR_TOWNS; j++) {
            if (d[i * NBR_TOWNS + j] == 0) {
                if (penaltys[i * NBR_TOWNS + j] > max_penalty) {
                    max_penalty = penaltys[i * NBR_TOWNS + j];
                    izero = i;
                    jzero = j;
                }
            }
        
        }
    }
    // printf("debug\n");

    if (izero < 0 || jzero < 0) {
        return;
    }
    next_town[izero] = jzero;

    /* Do the modification on a copy of the distance matrix */
    float d2[NBR_TOWNS][NBR_TOWNS];
    memcpy(d2, d, NBR_TOWNS * NBR_TOWNS * sizeof(float));

    for (k = 0; k < NBR_TOWNS; k++) {
        d2[izero][k] = -1;
        d2[k][jzero] = -1;
    }
    d2[jzero][izero] = -1; // si B -> A alors A -> B interdit

    /* Explore left child node according to given choice */

    little_algorithm(d2, iteration + 1, eval_node_child);

    /* Do the modification on a copy of the distance matrix */
    memcpy(d2, d, NBR_TOWNS * NBR_TOWNS * sizeof(float));

    // next_town[izero] = -1;
    d2[izero][jzero] = -1;

    little_algorithm(d2, iteration, eval_node_child + max_penalty);

    hipFree(d);
}


int main(int argc, char* argv[]) {
    printf("Little Algorithm\n");
#ifdef LITTLEPLUS
    printf("LITTLEPLUS is defined\n");
#else
    printf("LITTLEPLUS is not defined\n");
#endif
    printf("Number of towns: %d\n", NBR_TOWNS);
    for (int i = 0; i < NBR_TOWNS; i++)
        next_town[i] = -1;
    
    best_eval = -1;

    compute_matrix(dist);

    /* initial solution */
    float initial_value = initial_solution();
    (void)initial_value;

    /** Little : uncomment when needed
    */
    int iteration = 0;
    float lowerbound = 0.0;

    time_t start = time(NULL);
    little_algorithm(dist, iteration, lowerbound);
    time_t end = time(NULL);


    printf("Best solution:");
    print_solution(best_solution, best_eval);

    printf("Time: %f seconds\n", difftime(end, start));
    return 0;
}
